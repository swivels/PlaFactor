#include "hip/hip_runtime.h"
#include <cassert>
#include "pla.h"

template<typename numT>
oneDArray<numT>::oneDArray(const oneDArray<numT> &other) {
    rows = other.rows;
    cols = other.cols;
    size = other.size;
    alrows = other.alrows;
    alcols = other.alcols;
    alsize = other.alsize;
    parent = other.parent;

    // Allocate memory for the array and copy the data
    arr = new numT[alsize];
    memcpy(arr, other.arr, other.alsize * sizeof(numT));
}
template oneDArray<uint8_t>::oneDArray(const oneDArray<uint8_t> &other);
template oneDArray<uint32_t>::oneDArray(const oneDArray<uint32_t> &other);

template <typename numT>
oneDArray<numT>::~oneDArray() {
    delete[] arr;
}
template oneDArray<uint8_t>::~oneDArray();
template oneDArray<uint32_t>::~oneDArray();

template <typename numT>
void oneDArray<numT>::initArr() {
    size = rows * cols;
    alsize = alrows * alcols;
    arr = new numT[alsize];
    memset(arr,0,alsize*sizeof(numT)); //todo: maybe make this uint8_t instead of numT bc memset only works with bytes
}
template void oneDArray<uint8_t>::initArr();
template void oneDArray<uint32_t>::initArr();

template <typename numT>
NVCC_BOTH numT* oneDArray<numT>::ind_ptr(index_t row, index_t col) {
    return arr+ind(row,col);
}
template NVCC_BOTH uint8_t* oneDArray<uint8_t>::ind_ptr(index_t row, index_t col);
template NVCC_BOTH uint32_t* oneDArray<uint32_t>::ind_ptr(index_t row, index_t col);

// indexing function which returns 1d index
template <typename numT>
NVCC_BOTH index_t oneDArray<numT>::ind(index_t row, index_t col) const { //todo: make inline and move to .h file for speed, do profiling
    if (row >= alrows || col >= alcols) //todo: maybe remove this check for speed
        OnGpuErr("arr index is out of allocated bounds--ind function");
    return alcols * row + col;
}
template NVCC_BOTH index_t oneDArray<uint8_t>::ind(index_t row, index_t col) const;
template NVCC_BOTH index_t oneDArray<uint32_t>::ind(index_t row, index_t col) const;

template <>
NVCC_BOTH index_t oneDArray<uint32_t>::indOrdered(index_t row, index_t col) const { //for count array only
    if (row >= alrows || col >= alcols)
        OnGpuErr("arr index is out of allocated bounds--indOrdered function");
    if (row>col)
        return alcols * col + row;
    else
        return alcols * row + col;
}
template NVCC_BOTH index_t oneDArray<uint32_t>::indOrdered(index_t row, index_t col) const;

template <typename numT>
NVCC_BOTH index_pair oneDArray<numT>::indTwoD(index_t index) const {
    if (index>=alsize) //todo: maybe remove this check for speed
        OnGpuErr("arr index is out of allocated bounds--indTwoD function");
    uint32_t row = index/alcols; //int row = index/alcols;
    uint32_t col = index%alcols; //int col = index%alcols;
    if(row>=rows||col>=cols)
        OnGpuErr("index is out of used bounds--indTwoD function");
    return {row,col};//index_pair(row,col);//make_pair(row,col);
}
template NVCC_BOTH index_pair oneDArray<uint8_t>::indTwoD(index_t index) const;
template NVCC_BOTH index_pair oneDArray<uint32_t>::indTwoD(index_t index) const;

// indexing function which returns value
template <typename numT>
NVCC_BOTH numT oneDArray<numT>::get_val(index_t row, index_t col) const { //todo: make inline and move to .h file for speed, do profiling
    if (row >= alrows || col >= alcols) //todo: maybe remove this check for speed
        OnGpuErr("arr index is out of allocated bounds--get_val function");
    return arr[alcols * row + col];
}
template NVCC_BOTH uint8_t oneDArray<uint8_t>::get_val(index_t row, index_t col) const;
template NVCC_BOTH uint32_t oneDArray<uint32_t>::get_val(index_t row, index_t col) const;

template <typename numT>
NVCC_BOTH void oneDArray<numT>::set_val(index_t row, index_t col, numT val) const {
    if (row >= alrows || col >= alcols) //todo: maybe remove this check for speed
        OnGpuErr("arr index is out of allocated bounds, cannot set value at this index--set_val function");
    arr[alcols * row + col] = val;
}
template NVCC_BOTH void oneDArray<uint8_t>::set_val(index_t row, index_t col, uint8_t val) const;
template NVCC_BOTH void oneDArray<uint32_t>::set_val(index_t row, index_t col, uint32_t val) const;

template <>
NVCC_BOTH void oneDArray<uint8_t>::printRow(index_t row) const {
    for (int i = 0; i < cols; i++)
        printf("%c",'0' + get_val(row, i));
}
template NVCC_BOTH void oneDArray<uint8_t>::printRow(index_t row) const;

template <>
NVCC_BOTH void oneDArray<uint32_t>::printRow(index_t row) const {
    for (int i = 0; i < cols; i++) {
        if(i <= row) {
            printf("-  ");
            continue;
        }
        uint32_t number = get_val(row, i);
        int count = floor(log10((double)number)) + 1;
        printf("%d%s", number, (count>1?" ":"  "));
    }
}
template NVCC_BOTH void oneDArray<uint32_t>::printRow(index_t row) const;

template <>
NVCC_BOTH void oneDArray<uint8_t>::printArr() const {
    for (index_t row = 0; row < alrows; row++) {
        if (!parent->usedrows[row])
            continue;
        printRow(row);
        printf("\n");
    }
}
template NVCC_BOTH void oneDArray<uint8_t>::printArr() const;

template <>
NVCC_BOTH void oneDArray<uint32_t>::printArr() const {
    for (index_t row = 0; row < rows; row++) {
        printRow(row);
        printf("\n");
    }
}
template NVCC_BOTH void oneDArray<uint32_t>::printArr() const;

template <typename numT>
vector<string> oneDArray<numT>::convertBackFormat() {
    /*
    returns a vector of strings for each row, converted back into standard PLA format,
    re-adding the removed - and halfing the number of literals
    */
    vector<string> toReturn;
    for (index_t row = 0; row < alrows; row++) {
        if (!parent->usedrows[row])
            continue;
        string line;
        index_t start = ind(row, 0);
        for (index_t col = 0; col < cols; col=col+2) {
            if (arr[start + col] == 0 && arr[start + col + 1] == 0) {
                line.push_back('-');
            }
            else if (arr[start + col] == 1) {
                line.push_back('0');
            }
            else if (arr[start + col + 1] == 1) {
                line.push_back('1');
            }
        }
        toReturn.push_back(line);
    }
    return toReturn;
}
template vector<string> oneDArray<uint8_t>::convertBackFormat();
template vector<string> oneDArray<uint32_t>::convertBackFormat();

template <typename numT>
vector<string> oneDArray<numT>::convertToVecString() {
    //simply converts arr to a vector of strings with no additional modification
    vector<string> toReturn;
    for (index_t row = 0; row < alrows; row++) {
        if (!parent->usedrows[row])
            continue;
        string line;
        index_t start = ind(row, 0);
        for (index_t col = 0; col < cols; col++) {
            line.push_back('0' + arr[start + col]);
        }
        toReturn.push_back(line);
    }
    return toReturn;
}
template vector<string> oneDArray<uint8_t>::convertToVecString();
template vector<string> oneDArray<uint32_t>::convertToVecString();

template<>
NVCC_BOTH index_t oneDArray<uint32_t>::findBiggestWeight(int minimumGain) const{
    //finds the biggest weight in the table

    index_t indMax,i;
    indMax = 0;
    i = 0;
    int temp,max;
    temp = 0;
    max = 0;
    printf("Finding the biggest weight...\n");
    for (; i < alsize; i++) {
        temp = arr[i];
        if (temp >rows) //rows used to be passed in as itable.rows
            continue;
        if(temp>max) {
            max = temp;
            indMax = i;
        }
    }
    if((max < 2-minimumGain))
        return alsize;
    return indMax;
}
template NVCC_BOTH index_t oneDArray<uint32_t>::findBiggestWeight(int minCost) const;

template <typename numT>
void oneDArray<numT>::retrieveDataGpu(oneDArray<numT> *Gpup) {
    //copies data from GPU to CPU
    //todo: check if sizes match before copying and reallocate on host if needed
    GpuMemcpyDeviceToHost(arr, Gpup->arr, alsize * sizeof(numT));
}
template void oneDArray<uint8_t>::retrieveDataGpu(oneDArray<uint8_t> *Gpup);
template void oneDArray<uint32_t>::retrieveDataGpu(oneDArray<uint32_t> *Gpup);